#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>((&pointer))[0])

// used for
struct __align__(8) MS {float m; float s;};

__device__ MS warpReduce(MS val)
{
    for(int offset = 16; offset > 0; offset /= 2)
    {
        MS tmp;
        tmp.m = __shfl_xor_sync(0xFFFFFFFF, val.m, offset);
        tmp.s = __shfl_xor_sync(0xFFFFFFFF, val.s, offset);

        bool is_val_bigger = val.m > tmp.m;
        MS bigger = is_val_bigger ? val : tmp;
        MS smaller = is_val_bigger ? tmp : val;

        val.m = bigger.m;
        val.s = bigger.s + smaller.s * expf(smaller.m - bigger.m); // 更小的项需要添加偏移量
    }
    return val;
}

__device__ MS reduce2MS(MS x, MS y)
{
    bool is_bigger = x.m > y.m;
    MS bigger = is_bigger ? x : y;
    MS smaller = is_bigger ? y : x;
    
    MS tmp;
    tmp.m = bigger.m;
    tmp.s = bigger.s + smaller.s * expf(smaller.m - bigger.m);
    
    return tmp;
}
__global__ void online_softmax(float* data, float* out, int N, int C)
{
    // oneline softmax kernel
    // The shape of data is N * C
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    MS thread_res = {-INFINITY, 0}; 
    
    while(tid < C)
    {
        float new_val = data[bid * C + tid];
        float new_max_val = fmax(thread_res.m, new_val);
        thread_res.s = thread_res.s * expf(thread_res.m - new_max_val) + expf(new_val - new_max_val);
        thread_res.m = new_max_val;
        tid += blockDim.x;
    }

    __shared__ MS warpRes[32];
    MS thread_res = warpReduce(thread_res);
    if(tid % 32 == 0) {
        warpRes[tid / 32] = thread_res;
    }
    __syncthreads();
    MS empty = {0, 0};
    MS thread_res = tid < 32 ? warpRes[tid] : empty;
    MS block_res = warpReduce(thread_res);

    __shared__ MS blockRes;
    if(tid == 0) {
        // every block write back the res to smem
        blockRes = block_res;
    }

    __syncthreads();
    // the final pass
    for(int i = tid; i < C; i += blockDim.x)
    {
        int idx = bid * C + tid;
        out[idx] = expf(data[idx] - blockRes.m) / blockRes.s;
    }
}

template<int BLOCK_SIZE>  // assume the BLOCK_SIZE is 32
__global__ void transpose(float* data, float* out, int M, int N)
{
    // transpose kernel uses smem and avoid bank conflict
    // The shape of input matrix is M * N
    // The shape of output matrix is N * M
    // grid [N / BLOCKSIZE, M / BLOCKSIZE]  block [BLOCK_SIZE, BLOCK_SIZE]
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    __shared__ float smem[BLOCK_SIZE][BLOCK_SIZE + 1]; // padding 
    int row_A = by * blockDim.y + ty;
    int col_A = bx * blockDim.x + tx;
    if(row_A < M && col_A < N) {
        smem[ty][tx] = data[row_A * N + col_A]; // 合并访问
    }
    
    int row_B = bx * blockDim.x + ty;
    int col_B = by * blockDim.y + tx;
    if(row_B < N && col_B < M) {
        data[row_B * M + col_B] = smem[tx][ty]; // 合并访问
    }
}

template <
const int BLOCK_SIZE_M,
const int BLOCK_SIZE_K,
const int BLOCK_SIZE_N
>
__global__ void gemm_v1(
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K)
{
    // a tiling gemm kernel 
    // A: M * K   B: K * N   C: M * N 
    // a block is responsible for a small matrix with BLOCK_SIZE_M * BLOCK_SIZE_N

    // block: (BLOCK_SIZE_M, BLOCK_SIZE_N)  grid: (M / BLOCK_SIZE_M, N / BLOCK_SIZE_N)

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = tx * blockDim.y + ty;
    __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_M];

    int ITER_PER_BLOCK = K / BLOCK_SIZE_K; // 一个block要做多少次循环

    A = &A[BLOCK_SIZE_M * bx * K];
    B = &B[BLOCK_SIZE_N * by]; 
    float final_thread_res = 0.0f;
    for(iter = 0; iter < ITER_PER_BLOCK; iter ++) 
    {
        // 每次循环计算部分结果，所有部分结果相加即为最终结果
        
        // 1. load the tiling matrix to As and Bs
        // assume that BM * BN > BM * BK and BN * BK
        // one thread is responsible for a element loading

        // load As
        if(tid < BLOCK_SIZE_M * BLOCK_SIZE_K) {
            int row_id = tid / BLOCK_SIZE_K;
            int col_id = tid % BLOCK_SIZE_K;
            As[row_id][col_id] = A[row_id * K + iter * BLOCK_SIZE_K + col_id];
        }

        // load Bs
        if(tid < BLOCK_SIZE_N * BLOCK_SIZE_K) {
            int row_id = tid / BLOCK_SIZE_M;
            int col_id = tid % BLOCK_SIZE_M;
            Bs[row_id][col_id] = B[iter * BLOCK_SIZE_K * N + row_id * N + col_id];
        }

        __syncthreads();

        // do thread-level computation
        float thread_res = 0.0f;
        for(int i = 0; i < BLOCK_SIZE_K; i ++) {
            thread_res += As[tx][i] * Bs[i][ty];
        }

        // write back the res
        final_thread_res += thread_res;
    }

    // write back to Matrix C
    int row_c = tx * BLOCK_SIZE_M + tx;
    int col_c = ty * BLOCK_SIZE_N + ty;
    C[row_c * N + col_c] = final_thread_res;

    // done!
}