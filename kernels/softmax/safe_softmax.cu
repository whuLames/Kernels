#include "hip/hip_runtime.h"
// The implementation of the softmax function with a safe softmax trick.
#include <stdio.h>
#include <hip/hip_runtime.h>

// the struct for online softmax
struct __align__(8) MS {float m; float s;};

__device__ MS onlineWarpReduce(MS val)
{
    #pragma unroll
    for(int offset=16; offset>0; offset>>=1) {
        MS tmp;
        tmp.m = __shfl_xor_sync(0xFFFFFFFF, val.m, offset);
        tmp.s = __shfl_xor_sync(0xFFFFFFFF, val.s, offset);
        bool val_bigger = val.m > tmp.m;
        MS bigger = val_bigger ? val : tmp;
        MS smaller = val_bigger ? tmp : val;
        val.m = bigger.m;
        val.s = smaller.s * expf(bigger.m - smaller.m) + bigger.s; 
    }

    return val;
}

__device__ float atomicMax(float* address, float val)
{
    int* address_as_i = (int*)address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while(assumed != old);
    return __int_as_float(old);
}

__device__ float warpMaxReduce(float val)
{
    #pragma unroll
    for(int offset=16; offset>0; offset>>=1) {
        val = max(val, __shfl_xor_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}
__device__ float warpSumReduce(float val)
{
    #pragma unroll
    for(int offset=16; offset>0; offset>>=1) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ float blockMaxReduce(float val)
{
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    __shared__ float warpRes[64];
    val = warpMaxReduce(val);
    if(lane_id==0) warpRes[warp_id] = val;
    __syncthreads();
    int tid = threadIdx.x;

    val = tid < 32 ? warpRes[tid] : 0;

    return warpMaxReduce(val);
}

__device__ float blockSumReduce(float val)
{
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    __shared__ float warpRes[64];
    val = warpSumReduce(val);
    if(lane_id==0) warpRes[warp_id] = val;
    __syncthreads();
    int tid = threadIdx.x;

    val = tid < 32 ? warpRes[tid] : 0;
    return warpSumReduce(val);
}



__global__ void safe_softmax(float* data, float* out, float* max_val, float* sum_val, int N)
{
    // Compute the max value
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    float val = data[bid * blockDim.x + tid]; 

    // get the max value of the current block
    float val_in_block = blockMaxReduce(val);

    if(tid == 0)
    {
        atomicMax(max_val, val_in_block);
    }
    __syncthreads();
    
    // Compute the sum of the exp value
    float val_exp = expf(val - *max_val);
    float sum_exp_in_block = blockSumReduce(val_exp);
    if(tid==0)
    {
        atomicAdd(out, sum_exp_in_block); 
    }
    __syncthreads();

    // Compute the softmax value
    out[bid * blockDim.x + tid] = val_exp / *sum_val;
}

__device__ MS reduce2MS(MS x, MS y)
{
    bool x_bigger = x.m > y.m;
    MS bigger = x_bigger ? x : y;
    MS smaller = x_bigger ? y : x;
    x.m = bigger.m;
    x.s = smaller.s * expf(bigger.m - smaller.m) + bigger.s;
    return x;
}
__global__ void online_softmax(float* data, float* out, int N, int C)
{
    // 2-d softmax and one block for a row
    // online softmax --> 2-pass
    // 1st pass: compute the max value and sum of the exp value
    // 2nd pass: compute the softmax value
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int lane_id = tid % 32;
    int warp_id = tid / 32;
    float max_val = -INFINITY;
    float sum_val = 0.0f;
    int stride = blockDim.x;
    MS val = {-INFINITY, 0.0f};
    __shared__ MS warpRes[32];
    __shared__ MS blockRes;
    for(int i = tid; i < C; i += stride)
    {
        float new_val = data[bid * C + i];
        float new_max_val = fmaxf(new_val, max_val);
        val.m = new_max_val;
        val.s = val.s * expf(new_max_val - max_val) + expf(new_val - val.m); // 前者修正 后者加上当前位置的value
    }
    MS res = onlineWarpReduce(val);
    if(lane_id == 0)
    {
        warpRes[warp_id] = res;
    }
    __syncthreads();

    MS val = tid < 32 ? warpRes[tid] : (MS){0.0f, 0.0f};
    MS block_res = onlineWarpReduce(val);
    if(tid == 0)
    {
        blockRes = block_res;
    }
    __syncthreads();

    // 2nd pass
    for(int i = tid; i < C; i += stride)
    {
        out[bid * C + i] = expf(data[bid * C + i] - blockRes.m) / blockRes.s;
    }

}
int main(int argc, char** argv)
{
    const int N = atoi(argv[1]);
    // printf("N = %d\n", N);
    float* h_data = (float*)malloc(N * sizeof(float));
    float* d_data;
    for(int i = 0; i < N; i++)
    {
        h_data[i] = i % 16;
    }
    hipMalloc((void**)&d_data, N * sizeof(float));
    hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(1024);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    float* d_out;
    float* d_max_val;
    float* d_sum_val;
    hipMalloc((void**)&d_out, N * sizeof(float));
    hipMalloc((void**)&d_max_val, sizeof(float));
    hipMalloc((void**)&d_sum_val, sizeof(float));
    hipMemset(d_out, 0, N * sizeof(float));
    hipMemset(d_max_val, 0, sizeof(float));
    hipMemset(d_sum_val, 0, sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    safe_softmax<<<gridSize, blockSize>>>(d_data, d_out, d_max_val, d_sum_val, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds);

}