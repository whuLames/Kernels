#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
#define THREAD_NUM_PER_BLOCK 256 


// The kernel function of one-iter reduce
__global__ void reduce_0(float* in, float* out)
{
    // shared memory
    __shared__ float sdata[THREAD_NUM_PER_BLOCK];

    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x + tid;

    // load data to shared memory
    sdata[tid] = in[id];
    __syncthreads();

    // reduce
    for(int s = 1; s < blockDim.x; s *= 2)
    {
        // if(tid % (2 * s) == 0)
        // {
        //     sdata[tid] += sdata[tid + s];
        // }

        if((tid&(2*s - 1)) == 0) // 这里如何理解
        {
            sdata[tid] += sdata[tid + s];
        }

        // if(tid % (2*s) == 0) 
        // {
        //     sdata[tid] += sdata[tid + s];
        // }
        __syncthreads();
    }

    // write the result to global memory
    if(tid == 0) out[blockIdx.x] = sdata[0];
}

int main(int argc, char** argv)
{
    // The number of element to reduce
    int N = argc == 2 ? atoi(argv[1]) : 32 * 1024 * 1024;
    cout << "N: " << N << endl;
    int blockNum = (N + THREAD_NUM_PER_BLOCK - 1) / THREAD_NUM_PER_BLOCK;
    cout << "blockNum: " << blockNum << endl;
    // memory malloc
    float* inCPU = (float*)malloc(N * sizeof(float));
    float* outCPU = (float*)malloc(blockNum * sizeof(float));
    float* inGPU;
    float* outGPU;

    hipMalloc((void**)&outGPU, blockNum * sizeof(float));
    hipMalloc((void**)&inGPU, N * sizeof(float));
    
    // data init
    for(int i = 0; i < N; i ++) inCPU[i] = 1.0f;
    
    float* ans = (float*)malloc(blockNum * sizeof(float));

    for(int i = 0; i < blockNum; i ++) ans[i] = 0.0f;
    // build the reduce result for correctness check
    for(int i = 0; i < blockNum; i ++)
    {
        for(int j = 0; j < THREAD_NUM_PER_BLOCK; j ++)
        {
            int idx = i * THREAD_NUM_PER_BLOCK + j;
            if(idx < N) ans[i] += inCPU[idx];
        }
    }

    // memory copy
    hipMemcpy(inGPU, inCPU, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 gridSize(blockNum, 1);
    dim3 blockSize(THREAD_NUM_PER_BLOCK, 1);

    // time record
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // computation 
    reduce_0<<<gridSize, blockSize>>>(inGPU, outGPU);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "The number of reduced element: " << N << " Elapsed Time: " << elapsedTime << "ms" << endl;


    // memory copy
    hipMemcpy(outCPU, outGPU, blockNum * sizeof(float), hipMemcpyDeviceToHost);

    // check the correctness
    for(int i = 0; i < blockNum; i ++)
    {
        if(outCPU[i] != ans[i])
        {
            cout << "Error: " << i << " " << outCPU[i] << " " << ans[i] << endl;
        }
    }

    return 0;

}